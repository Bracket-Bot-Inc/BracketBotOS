// CUDA port of update_logodds_hash + clamp_logodds
// Suitable for PyCUDA's SourceModule


#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>   // CUDART_INF_F
#include <math_functions.h>   // isfinite, expf, rintf, etc.

#define EMPTY_KEY64 0xffffffffffffffffULL

// pack signed 21 bits/axis: [-2^20 .. 2^20-1]
__device__ __forceinline__
unsigned long long voxel_key64(long long i, long long j, long long k) {
    return (((unsigned long long)(i & 0x1FFFFF)) << 42) |
           (((unsigned long long)(j & 0x1FFFFF)) << 21) |
           (((unsigned long long)(k & 0x1FFFFF)) <<  0);
}

__device__ __forceinline__
unsigned long long hash64_u64(unsigned long long x, unsigned long long M) {
    x ^= x >> 33; x *= 0xff51afd7ed558ccdULL;
    x ^= x >> 33; x *= 0xc4ceb9fe1a85ec53ULL;
    x ^= x >> 33;
    return x % M;
}

extern "C" __global__
void update_logodds_hash(
    const float4 * __restrict__ origins,
    const float4 * __restrict__ endpoints,
    const float voxel_size,
    const int   max_steps,
    unsigned long long * __restrict__ keys,   // 64-bit keys
    int   * __restrict__ values,              // int32 log-odds
    const unsigned long long M,
    const int  hit_inc,
    const int  miss_dec,
    const int  n_valid,
    const float decay_lambda,
    const float min_hit_scale
){
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= n_valid) return;

    const float3 o = make_float3(origins[gid].x,  origins[gid].y,  origins[gid].z);
    const float3 e = make_float3(endpoints[gid].x, endpoints[gid].y, endpoints[gid].z);

    const float3 v = make_float3(e.x - o.x, e.y - o.y, e.z - o.z);
    const float   L = sqrtf(v.x*v.x + v.y*v.y + v.z*v.z);
    if (!isfinite(L) || L <= 0.0f) return;
    const float3 d = make_float3(v.x / L, v.y / L, v.z / L);

    int steps_needed = (int)ceilf(L / voxel_size) + 1;
    if (steps_needed > max_steps) steps_needed = max_steps;

    long long ix = (long long)floorf(o.x / voxel_size);
    long long iy = (long long)floorf(o.y / voxel_size);
    long long iz = (long long)floorf(o.z / voxel_size);

    const long long ix_end = (long long)floorf(e.x / voxel_size);
    const long long iy_end = (long long)floorf(e.y / voxel_size);
    const long long iz_end = (long long)floorf(e.z / voxel_size);

    const int step_x = (d.x > 0.0f) ? 1 : -1;
    const int step_y = (d.y > 0.0f) ? 1 : -1;
    const int step_z = (d.z > 0.0f) ? 1 : -1;

    float tMaxX = (d.x == 0.0f) ? HIP_INF_F :
        ((step_x > 0 ? ((ix+1)*voxel_size - o.x) : (ix*voxel_size - o.x)) / d.x);
    float tMaxY = (d.y == 0.0f) ? HIP_INF_F :
        ((step_y > 0 ? ((iy+1)*voxel_size - o.y) : (iy*voxel_size - o.y)) / d.y);
    float tMaxZ = (d.z == 0.0f) ? HIP_INF_F :
        ((step_z > 0 ? ((iz+1)*voxel_size - o.z) : (iz*voxel_size - o.z)) / d.z);

    const float tDeltaX = (d.x == 0.0f) ? HIP_INF_F : voxel_size / fabsf(d.x);
    const float tDeltaY = (d.y == 0.0f) ? HIP_INF_F : voxel_size / fabsf(d.y);
    const float tDeltaZ = (d.z == 0.0f) ? HIP_INF_F : voxel_size / fabsf(d.z);

    // free-space updates
    for (int s = 0; s < steps_needed; ++s) {
        if (ix == ix_end && iy == iy_end && iz == iz_end) break;

        const unsigned long long k = voxel_key64(ix, iy, iz);
        unsigned long long h = hash64_u64(k, M);
        // linear probing with a small cap (64)
        for (int attempt = 0; attempt < 64; ++attempt) {
            const unsigned long long slot = (h + (unsigned long long)attempt) % M;
            const unsigned long long prev = atomicCAS(&keys[slot], (unsigned long long)EMPTY_KEY64, k);
            if (prev == EMPTY_KEY64 || prev == k) {
                atomicAdd(&values[slot], (int)miss_dec);
                break;
            }
        }

        if (tMaxX < tMaxY && tMaxX < tMaxZ) { ix += step_x; tMaxX += tDeltaX; }
        else if (tMaxY < tMaxZ)             { iy += step_y; tMaxY += tDeltaY; }
        else                                { iz += step_z; tMaxZ += tDeltaZ; }
    }

    // endpoint with distance-based decay
    {
        float scale = expf(-L / fmaxf(decay_lambda, 1e-6f));
        if (scale < min_hit_scale) scale = min_hit_scale;
        const int scaled_hit = (int)rintf((float)hit_inc * scale);

        const unsigned long long k = voxel_key64(ix_end, iy_end, iz_end);
        unsigned long long h = hash64_u64(k, M);
        for (int attempt = 0; attempt < 64; ++attempt) {
            const unsigned long long slot = (h + (unsigned long long)attempt) % M;
            const unsigned long long prev = atomicCAS(&keys[slot], (unsigned long long)EMPTY_KEY64, k);
            if (prev == EMPTY_KEY64 || prev == k) {
                atomicAdd(&values[slot], (int)scaled_hit);
                break;
            }
        }
    }
}

// Direct port (assumes launch size == len(values))
extern "C" __global__
void clamp_logodds(int *values, const int lo, const int hi) {
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    int v = values[gid];
    if (v < lo) values[gid] = lo;
    if (v > hi) values[gid] = hi;
}

// Safer variant with bound check (optional)
extern "C" __global__
void clamp_logodds_n(int *values, const int lo, const int hi, const int n) {
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= n) return;
    int v = values[gid];
    if (v < lo) values[gid] = lo;
    if (v > hi) values[gid] = hi;
}
